#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

extern "C" {

#include"cuda_functions.h"

}

#define N 3600
#define GENERATION 500
#define MAX_THREAD_NUM 1024

// Kernel that calculates each creatures new state, depending on its neighbors
__global__ void calc_gen(int *in, int *neigh, int *out) {
	extern __shared__ int sha[];
	// Divide extern for readability
	int *state = sha;

	int tid = threadIdx.x;
	int creature_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	// Calculate k (number of creatures in row)
	int k = sqrt((float) blockDim.x);

	// Save state of current creature
	state[tid] = in[creature_id];

	// Outside rows and columns
	// The total states are equal to the number of threads (creatures) in each
	// block (blockDim.x or k*k) so start the shared memory for the Outside
	// rows and columns after that
	int *up = &state[blockDim.x];
	int *down = &up[k];
	int *right = &down[k];
	int *left = &right[k];

	// Useful results for determining creature's position in each block
	int tid_div = tid / k;
	int tid_mod = tid % k;

	// Get outside rows and cols and save in shared
	// If its the upper row, then each upper thread (creature) stores the
	// neighboring creature that is up
	if (tid_div == 0) {
		up[tid_mod] = in[(neigh[blockIdx.x * 8] * blockDim.x) + blockDim.x - k + tid];
	}
	// If its the last row, then each thread (creature) stores the
	// neighboring creature that is down
	else if (tid_div == k - 1) {
		down[tid_mod] = in[(neigh[blockIdx.x * 8 + 1] * blockDim.x) + tid];
	}
	// If its the left column, then each thread (creature) stores the
	// neighboring creature that is left
	else if (tid_mod == 0) {
		left[tid_div] = in[(neigh[blockIdx.x * 8 + 2] * blockDim.x) + (tid_div*k + k - 1)];
	}
	// If its the right column, then each thread (creature) stores the
	// neighboring creature that is right
	else if (tid_mod == k - 1) {
		right[tid_div] = in[(neigh[blockIdx.x * 8 + 3] * blockDim.x) + (tid_div * k)];
	}

	int neighbor_count = 0;
	// Synchronise so that shared memory is all there
	__syncthreads();

	// Calculate next creature state
	if (tid_div == 0) {
		// For the upper left creature
		if (tid == 0) {
			int up_left = in[(neigh[blockIdx.x * 8 + 4] * blockDim.x) + k*k - 1];
			neighbor_count = up_left + up[0] + up[1] + left[0] + left[1] + state[1]
							+ state[k] + state[k + 1];
		}
		// For the upper right creature
		else if (tid == k - 1) {
			int up_right = in[(neigh[blockIdx.x * 8 + 5] * blockDim.x) + k * (k - 1)];
			neighbor_count = up_right + up[tid_mod] + up[tid_mod - 1] + right[0] + right[1]
							+ state[tid - 1] + state[tid + k - 1] + state[tid + k];
		}
		// For the upper row
		else {
			neighbor_count = up[tid_mod - 1] + up[tid_mod] + up[tid_mod + 1] + state[tid - 1]
							+ state[tid + 1] + state[tid + k - 1] + state[tid + k]
							+ state[tid + k + 1];
		}
	}
	else if (tid_div == k - 1) {
		// For the downmost left creature
		if (tid == k * (k - 1)) {
			int down_left = in[(neigh[blockIdx.x * 8 + 6] * blockDim.x) + k - 1];
			neighbor_count = down_left + left[tid_div - 1] + left[tid_div] + down[0] + down[1]
							+ state[tid + 1] + state[tid - k] + state[tid - k + 1];
		}
		// For the downmost right creature
		else if (tid == k*k - 1) {
			int down_right = in[(neigh[blockIdx.x * 8 + 7] * blockDim.x)];
			neighbor_count = down_right + right[tid_div - 1] + right[tid_div]
							+ down[k - 1] + down[k - 2] + state[tid - 1]
							+ state[tid - k] + state[tid - k - 1];
		}
		// For the downmost row
		else {
			neighbor_count = down[tid_mod - 1] + down[tid_mod] + down[tid_mod + 1] +
							+ state[tid - 1] + state[tid + 1]
							+ state[tid - k] + state[tid - k - 1] + state[tid - k + 1];
		}
	}
	// For the leftmost column
	else if (tid_mod == 0) {
		neighbor_count = left[tid_div] + left[tid_div - 1] + left[tid_div + 1]
						+ state[tid + 1] + state[tid - k] + state[tid - k + 1]
						+ state[tid + k] + state[tid + k + 1];
	}
	// For the rightmost column
	else if (tid_mod == k - 1) {
		neighbor_count = right[tid_div] + right[tid_div - 1] + right[tid_div + 1]
						+ state[tid - 1] + state[tid - k] + state[tid - k - 1]
						+ state[tid + k] + state[tid + k - 1];
	}
	// Normally inside the perimeter
	else {
		neighbor_count = state[tid - k - 1] + state[tid - k] + state[tid - k + 1]
						+ state[tid - 1] + state[tid + 1]
						+ state[tid + k - 1] + state[tid + k] + state[tid + k + 1];
	}

	// Finally calculate the new state of the creature
	if (state[tid] == 1) {
		if ((neighbor_count == 2) || (neighbor_count == 3)) {
			out[creature_id] = 1;
		}
		else {
			out[creature_id] = 0;
		}
	}
	// Else if state[tid] == 0
	else {
		if (neighbor_count == 2) {
			out[creature_id] = 1;
		}
		else {
			out[creature_id] = 0;
		}
	}
}

// Same kernel with added shared memory to check if there is no change between
// generations (called every 20 generations, heavier)
// Contains a form of reduction as a means to quickly determine if
// a change has been made in the world
// The changed data for each block is stored in *chagned
__global__ void calc_gen_wcheck(int *in, int *neigh, int *out, int *changed) {
	extern volatile __shared__ int sha_wcheck[];
	// Divide extern for readability
 	volatile int *state = sha_wcheck;

	int tid = threadIdx.x;
	int creature_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	// Calculate k (number of creatures in row)
	int k = sqrt((float) blockDim.x);

	// Save state of current creature
	state[tid] = in[creature_id];

	// Outside rows and columns
	// The total states are equal to the number of threads (creatures) in each
	// block (blockDim.x or k*k) so start the shared memory for the Outside
	// rows and columns after that
	volatile int *up = &state[blockDim.x];
 	volatile int *down = &up[k];
 	volatile int *right = &down[k];
 	volatile int *left = &right[k];

	// Useful results for determining creature's position in each block
	int tid_div = tid / k;
	int tid_mod = tid % k;

	// Get outside rows and cols and save in shared
	// If its the upper row, then each upper thread (creature) stores the
	// neighboring creature that is up
	if (tid_div == 0) {
		up[tid_mod] = in[(neigh[blockIdx.x * 8] * blockDim.x) + blockDim.x - k + tid];
	}
	// If its the last row, then each thread (creature) stores the
	// neighboring creature that is down
	else if (tid_div == k - 1) {
		down[tid_mod] = in[(neigh[blockIdx.x * 8 + 1] * blockDim.x) + tid];
	}
	// If its the left column, then each thread (creature) stores the
	// neighboring creature that is left
	else if (tid_mod == 0) {
		left[tid_div] = in[(neigh[blockIdx.x * 8 + 2] * blockDim.x) + (tid_div*k + k - 1)];
	}
	// If its the right column, then each thread (creature) stores the
	// neighboring creature that is right
	else if (tid_mod == k - 1) {
		right[tid_div] = in[(neigh[blockIdx.x * 8 + 3] * blockDim.x) + (tid_div * k)];
	}

	int neighbor_count = 0;
	// Synchronise so that shared memory is all there
	__syncthreads();

	// Calculate next creature state
	if (tid_div == 0) {
		// For the upper left creature
		if (tid == 0) {
			int up_left = in[(neigh[blockIdx.x * 8 + 4] * blockDim.x) + k*k - 1];
			neighbor_count = up_left + up[0] + up[1] + left[0] + left[1] + state[1]
							+ state[k] + state[k + 1];
		}
		// For the upper right creature
		else if (tid == k - 1) {
			int up_right = in[(neigh[blockIdx.x * 8 + 5] * blockDim.x) + k * (k - 1)];
			neighbor_count = up_right + up[tid_mod] + up[tid_mod - 1] + right[0] + right[1]
							+ state[tid - 1] + state[tid + k - 1] + state[tid + k];
		}
		// For the upper row
		else {
			neighbor_count = up[tid_mod - 1] + up[tid_mod] + up[tid_mod + 1] + state[tid - 1]
							+ state[tid + 1] + state[tid + k - 1] + state[tid + k]
							+ state[tid + k + 1];
		}
	}
	else if (tid_div == k - 1) {
		// For the downmost left creature
		if (tid == k * (k - 1)) {
			int down_left = in[(neigh[blockIdx.x * 8 + 6] * blockDim.x) + k - 1];
			neighbor_count = down_left + left[tid_div - 1] + left[tid_div] + down[0] + down[1]
							+ state[tid + 1] + state[tid - k] + state[tid - k + 1];
		}
		// For the downmost right creature
		else if (tid == k*k - 1) {
			int down_right = in[(neigh[blockIdx.x * 8 + 7] * blockDim.x)];
			neighbor_count = down_right + right[tid_div - 1] + right[tid_div]
							+ down[k - 1] + down[k - 2] + state[tid - 1]
							+ state[tid - k] + state[tid - k - 1];
		}
		// For the downmost row
		else {
			neighbor_count = down[tid_mod - 1] + down[tid_mod] + down[tid_mod + 1] +
							+ state[tid - 1] + state[tid + 1]
							+ state[tid - k] + state[tid - k - 1] + state[tid - k + 1];
		}
	}
	// For the leftmost column
	else if (tid_mod == 0) {
		neighbor_count = left[tid_div] + left[tid_div - 1] + left[tid_div + 1]
						+ state[tid + 1] + state[tid - k] + state[tid - k + 1]
						+ state[tid + k] + state[tid + k + 1];
	}
	// For the rightmost column
	else if (tid_mod == k - 1) {
		neighbor_count = right[tid_div] + right[tid_div - 1] + right[tid_div + 1]
						+ state[tid - 1] + state[tid - k] + state[tid - k - 1]
						+ state[tid + k] + state[tid + k - 1];
	}
	// Normally inside the perimeter
	else {
		neighbor_count = state[tid - k - 1] + state[tid - k] + state[tid - k + 1]
						+ state[tid - 1] + state[tid + 1]
						+ state[tid + k - 1] + state[tid + k] + state[tid + k + 1];
	}

	// Finally calculate the new state of the creature
	int new_state = 0;
	int prev_state = state[tid];
	if (prev_state == 1) {
		if ((neighbor_count == 2) || (neighbor_count == 3)) {
			new_state = 1;
		}
		else {
			new_state = 0;
		}
	}
	// Prev_state == 0
	else {
		if (neighbor_count == 2) {
			new_state = 1;
		}
		else {
			new_state = 0;
		}
	}
	out[creature_id] = new_state;
	// Sync because the shared memory state will be repurposed
	__syncthreads();
	// Now use the state shared memory to save if the state of current creature has changed
	// Save 0 if the creature's state has not changed, 1 otherwise after calculating
	// the creature's new state
	if (new_state != prev_state) {
		state[tid] = 1;
	}
	else {
		state[tid] = 0;
	}
	// Begin custom reduction loop
	for(unsigned int s = blockDim.x/2; s > 32; s>>=1) {
		if (tid < s)
			state[tid] += state[tid + s];
		__syncthreads();
	}
	// Unroll last six iterations (same warp)
	if (tid < 32) {
		state[tid] += state[tid + 32];
		state[tid] += state[tid + 16];
		state[tid] += state[tid +  8];
		state[tid] += state[tid +  4];
		state[tid] += state[tid +  2];
		state[tid] += state[tid +  1];
	}
	// Write reduction result back to global memory
	if (tid == 0)
		changed[blockIdx.x] = state[0];
}

// Cuda kernel that calculates, for each block all its neighboring blocks (8)
// and outputs it in the out array
// (nearly the same code as the mpi and mpi_openmp implementations)
__global__ void calc_neighbors(int sq_of_P, int P_size , int *out) {
	int rank = (blockIdx.x * blockDim.x) + threadIdx.x;
	// Some threads will exceed the wanted rank outputs
	// (P_size % MAX_THREAD_NUM)
	if (rank < P_size) {

		int Pmod = rank % sq_of_P;
		int Pdiv = rank / sq_of_P;  //rank = div*sP+mod

		int D = ((Pdiv + 1) % sq_of_P)*sq_of_P + Pmod;
		int R  = Pdiv*sq_of_P + (Pmod + 1)%sq_of_P;
		int Dr = ((Pdiv + 1)%sq_of_P)*sq_of_P + (Pmod + 1)%sq_of_P;
		int U, L, Ul, Ur, Dl;

		if (Pdiv == 0) {
			U  = P_size - sq_of_P + Pmod;
			Ur = P_size - sq_of_P + (Pmod + 1)%sq_of_P;
			if (Pmod == 0)
				Ul = P_size - 1;
			else
				Ul = P_size - sq_of_P + Pmod - 1;
		}
		else {
			U  = (Pdiv - 1)*sq_of_P + Pmod;
			Ur = (Pdiv - 1)*sq_of_P + (Pmod + 1)%sq_of_P;
	    	if (Pmod == 0)
				Ul = Pdiv*sq_of_P - 1;
	        else
	        	Ul = (Pdiv - 1)*sq_of_P + Pmod - 1;
		}
		if (Pmod == 0) {
			L  = (Pdiv + 1)*sq_of_P - 1;
			Dl =((Pdiv + 1)%sq_of_P)*sq_of_P + sq_of_P - 1;
		}
		else {
			L  = Pdiv*sq_of_P + Pmod - 1;
			Dl = ((Pdiv + 1)%sq_of_P)*sq_of_P + Pmod - 1;
		}

		// Store results
		out[rank * 8] = U;
		out[rank * 8 + 1] = D;
		out[rank * 8 + 2] = L;
		out[rank * 8 + 3] = R;
		out[rank * 8 + 4] = Ul;
		out[rank * 8 + 5] = Ur;
		out[rank * 8 + 6] = Dl;
		out[rank * 8 + 7] = Dr;
	}
}

// Main
int main(int argc, char* argv[]) {
	// P_size is the number of blocks that the problem will be split into
	int P_size;
	if (argc >= 2)
		P_size = atoi(argv[1]);
	else
		return 1;

	int sqrt_P = sqrt(P_size);
	// sq_of_P=sqrt of P_size and k = N / sq_of_P (must be integers)
	int k  = N / sqrt_P;

	// k*k (creature number) should not be greater than the maximum thread number
	// of a block (depends of the gpu)
	if (k*k > MAX_THREAD_NUM) {
		printf("Please insert higher block number, so that each block has less than or equal to %d creatures \n", MAX_THREAD_NUM);
		return 1;
	}
	int world_datasize = N*N * sizeof(int);
	int *world = (int*)malloc(world_datasize);
	// Later used to check if world changed
	int *newWorld = (int*)malloc(world_datasize);

	// Create the world
	if (argc == 3) { //from text
		int *inArray = (int*)malloc(N*N * sizeof(int));
		getInputData(argv[4], inArray, N*N);
		makeArrayInBlocks(inArray, world, sqrt_P, k);
		free(inArray);
	}
	else
		makeRandomWorld(world, N*N);

	// Calculate neighbors
	int *d_neigh;
	hipMalloc((void**) &d_neigh, 8 * P_size * sizeof(int));
	// Divide problem use as many threads as possible
	calc_neighbors<<<MAX_THREAD_NUM, (P_size / MAX_THREAD_NUM) + 1>>>(sqrt_P, P_size, d_neigh);

	// Initialize main loop cuda memory
	int *d_world, *d_newWorld;
	hipMalloc((void**) &d_world, world_datasize);
	hipMalloc((void**) &d_newWorld, world_datasize);

	// A value stores 1 if the block that it represents has
	// changed, 0 otherwise
	int *changed = (int*)malloc(P_size * sizeof(int));
	for (int i = 0; i < P_size; i++)
		changed[i] = 0;

	int *d_changed;
	hipMalloc((void**) &d_changed, P_size * sizeof(int));
	hipMemcpy(d_changed, changed, P_size * sizeof(int), hipMemcpyHostToDevice);

	// Copy main world data
	hipMemcpy(d_world, &world, world_datasize, hipMemcpyHostToDevice);
	// Calculate shared memory size
	// sha_size = k*k (block creatures) + 4*k (4 neighboring rows and columns) * sizeof(int)
	int changed_flag = 0;
	for (int i = 0; i < GENERATION; i++) {
		// Check if world didn't change every 20 generations
		if( !((i+1) % 20) ) {
			// Mind the shared memory size
			calc_gen_wcheck <<<P_size, k*k, (k*k + 4*k) * sizeof(int)>>> (d_world, d_neigh, d_newWorld, d_changed);
			hipMemcpy(changed, d_changed, P_size * sizeof(int), hipMemcpyDeviceToHost);
			for (int ins = 0; ins < P_size; i++) {
				if (changed[ins] > 0) {
					changed_flag = 1;
					break;
				}
			}
			if(changed_flag == 0) {
				printf("Grid didn't change in %d generation! Exiting..\n", i + 1);
				break;
			}
			else {
				changed_flag = 0;
				// Reinitialise the d_changed array
				for (int i = 0; i < P_size; i++)
					changed[i] = 0;
				hipMemcpy(d_changed, changed, P_size * sizeof(int), hipMemcpyHostToDevice);
			}
		}
		// Else don't check or change global variables
		else {
			calc_gen <<<P_size, k*k, (k*k + 4*k) * sizeof(int)>>> (d_world, d_neigh, d_newWorld);
		}
		// Swap arrays for next generation iteration
		swapArrays(&d_world, &d_newWorld);
	}

	// Commented out, used for printing
	//hipMemcpy(newWorld, &d_newWorld, world_datasize, hipMemcpyDeviceToHost);
	// printworld(newWorld, sqrt_P, k);

	// Free memory
	free(world);
	free(newWorld);
	free(changed);
	// Free allocated cuda memory
	hipFree(d_neigh);
	hipFree(d_world);
	hipFree(d_newWorld);
	hipFree(d_changed);

	return 0;
}
